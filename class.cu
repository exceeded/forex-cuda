
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 200000

class BaseStrategy {
    private:
        double profitLoss;

    protected:
        __device__ void incrementProfitLoss() {
            this->profitLoss++;
        }

    public:
        __host__ BaseStrategy() {
            this->profitLoss = 0;
        }
        __device__ void backtest() {}
        __host__ double getProfitLoss() {
            return this->profitLoss;
        }
};

class Strategy : public BaseStrategy {
    public:
        __host__ Strategy() : BaseStrategy() {}
        __device__ void backtest() {
            incrementProfitLoss();
        }
};

__global__ void backtestStrategies(Strategy *strategies) {
    // Reference: https://devblogs.nvidia.com/parallelforall/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < N;
         i += blockDim.x * gridDim.x)
    {
        strategies[i].backtest();
    }
}

int main() {
    int blockCount = 32;
    int threadsPerBlock = 1024;

    Strategy *devStrategies;
    Strategy *strategies = (Strategy*)malloc(N * sizeof(Strategy));
    int i = 0;

    // Allocate memory for strategies on the GPU.
    hipMalloc((void**)&devStrategies, N * sizeof(Strategy));

    // Initialize strategies on host.
    for (i=0; i<N; i++) {
        strategies[i] = Strategy();
    }

    // Copy strategies from host to GPU.
    hipMemcpy(devStrategies, strategies, N * sizeof(Strategy), hipMemcpyHostToDevice);

    for (i=0; i<363598; i++) {
        backtestStrategies<<<blockCount, threadsPerBlock>>>(devStrategies);
    }

    // Copy strategies from the GPU.
    hipMemcpy(strategies, devStrategies, N * sizeof(Strategy), hipMemcpyDeviceToHost);

    // Display results.
    for (i=0; i<N; i++) {
        printf("%f\n", strategies[i].getProfitLoss());
    }

    // Free memory for the strategies on the GPU.
    hipFree(devStrategies);

    return 0;
}